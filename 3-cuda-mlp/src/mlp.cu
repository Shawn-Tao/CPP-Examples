#include "hip/hip_runtime.h"
#include <stdio.h>

#include "mlp.h"
#include "hip/hip_runtime.h"

void printCudaVersion()
{
    int runtime_ver, driver_ver;
    hipRuntimeGetVersion(&runtime_ver);
    hipDriverGetVersion(&driver_ver);
    printf("<<<=====================|||=====================>>>\n");
    printf("CUDA Compiled version: %d, %d, %d\n", __CUDACC_VER_MAJOR__, __CUDACC_VER_MINOR__, __CUDACC_VER_BUILD__);
    printf("CUDA Runtime version: %d\n", runtime_ver);
    printf("CUDA Driver version: %d\n", driver_ver);
    printf("<<<=====================|||=====================>>>\n\n");
}

__global__ void matrixMulKernel(float *matrixA, float *matrixB, float *matrixC, int rowsA, int colsA, int colsB)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rowsA && col < colsB)
    {
        float sum = 0.0f;
        for (int k = 0; k < colsA; k++)
        {
            sum += matrixA[row * colsA + k] * matrixB[k * colsB + col];
        }
        matrixC[row * colsB + col] = sum;

        // Print intermediate result
        // printf("Intermediate result at rowsA %d, colsB %d  [%d][%d]: %.5f\n", rowsA, colsB, row, col, sum);
    }
}

__global__ void matrixAddKernel(float *matrixA, float *matrixB, float *matrixC, int rows, int cols)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows && col < cols)
    {
        int index = row * cols + col;
        matrixC[index] = matrixA[index] + matrixB[index];
    }
}

__global__ void softmaxKernel(float *input, float *output, int rows, int cols)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows && col < cols)
    {
        int index = row * cols + col;

        // Compute the exponential of each element
        float expVal = expf(input[index]);

        // Compute the sum of exponentials for the row
        float sumExp = 0.0f;
        for (int i = 0; i < cols; ++i)
        {
            sumExp += expf(input[row * cols + i]);
        }

        // Compute the softmax value for the element
        output[index] = expVal / sumExp;
    }
}

// elu activation function
__global__ void eluKernel(float *input, float *output, int rows, int cols)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows && col < cols)
    {
        int index = row * cols + col;
        output[index] = input[index] > 0 ? input[index] : expf(input[index]) - 1;
    }
}

float *matrixMul(float *matrixA, float *matrixB, int rowsA, int colsA, int rowsB, int colsB)
{
    float *matrixC;

    printf("MUL: ROW A %d COLS A %d ROW B %d COLS B %d\n", rowsA, colsA, rowsB, colsB);

    hipMalloc((void **)&matrixC, rowsA * colsB * sizeof(float));

    dim3 blockSize(16, 16);
    dim3 gridSize((colsB + blockSize.x - 1) / blockSize.x, (rowsA + blockSize.y - 1) / blockSize.y);

    matrixMulKernel<<<gridSize, blockSize>>>(matrixA, matrixB, matrixC, rowsA, colsA, colsB);

    return matrixC;
}

float *matrixAdd(float *matrixA, float *matrixB, int rows, int cols)
{
    float *matrixC;

    printf("ADD: ROW %d COLS %d \n", rows, cols);

    hipMalloc((void **)&matrixC, rows * cols * sizeof(float));

    dim3 blockSize(16, 16);
    dim3 gridSize((cols + blockSize.x - 1) / blockSize.x, (rows + blockSize.y - 1) / blockSize.y);

    matrixAddKernel<<<gridSize, blockSize>>>(matrixA, matrixB, matrixC, rows, cols);

    return matrixC;
}

float *softmax(float *input, int rows, int cols)
{
    float *matrixC;

    printf(" ROW %d  COLS %d \n", rows, cols);

    hipMalloc((void **)&matrixC, rows * cols * sizeof(float));

    dim3 blockSize(16, 16);
    dim3 gridSize((cols + blockSize.x - 1) / blockSize.x, (rows + blockSize.y - 1) / blockSize.y);

    softmaxKernel<<<gridSize, blockSize>>>(input, matrixC, rows, cols);

    return matrixC;
}

float *elu(float *input, int rows, int cols)
{
    float *matrixC;

    printf("ELU: ROW %d COLS %d \n", rows, cols);

    hipMalloc((void **)&matrixC, rows * cols * sizeof(float));

    dim3 blockSize(16, 16);
    dim3 gridSize((cols + blockSize.x - 1) / blockSize.x, (rows + blockSize.y - 1) / blockSize.y);

    eluKernel<<<gridSize, blockSize>>>(input, matrixC, rows, cols);

    return matrixC;
}

MLP_Network::MLP_Network(int input_dim, int output_dim, std::vector<int> hidden_dim, std::vector<std::vector<float>> weight, std::vector<std::vector<float>> bias)
    : input_dim_(input_dim), output_dim_(output_dim), hidden_dim_(hidden_dim)
{
    weights_ = new float *[hidden_dim.size() + 1];
    biases_ = new float *[hidden_dim.size() + 1];
    for (int i = 0; i < hidden_dim.size() + 1; i++)
    {
        weights_[i] = new float[hidden_dim[i] * (i == 0 ? input_dim : hidden_dim[i - 1])];
        biases_[i] = new float[hidden_dim[i]];
    }

    //copy weights_ and bias to device
    for (int i = 0; i < hidden_dim.size(); i++)
    {
        hipMalloc(&weights_[i], hidden_dim[i] * (i == 0 ? input_dim : hidden_dim[i - 1]) * sizeof(float));
        hipMemcpy(weights_[i], weight[i].data(), hidden_dim[i] * (i == 0 ? input_dim : hidden_dim[i - 1]) * sizeof(float), hipMemcpyHostToDevice);
        hipMalloc(&biases_[i], hidden_dim[i] * sizeof(float));
        hipMemcpy(biases_[i], bias[i].data(), hidden_dim[i] * sizeof(float), hipMemcpyHostToDevice);
    }

    //output layer
    hipMalloc(&weights_[hidden_dim.size()], hidden_dim[hidden_dim.size() - 1] * output_dim * sizeof(float));
    hipMemcpy(weights_[hidden_dim.size()], weight[hidden_dim.size()].data(), hidden_dim[hidden_dim.size() - 1] * output_dim * sizeof(float), hipMemcpyHostToDevice);
    hipMalloc(&biases_[hidden_dim.size()], output_dim * sizeof(float));
    hipMemcpy(biases_[hidden_dim.size()], bias[hidden_dim.size()].data(), output_dim * sizeof(float), hipMemcpyHostToDevice);
}

MLP_Network::~MLP_Network(){
    for (int i = 0; i < hidden_dim_.size() + 1; i++)
    {
        hipFree(weights_[i]);
        hipFree(biases_[i]);
    }
}

void MLP_Network::forward(float *input, float *output)
{
    //memcpy input to device
    float *d_input;
    int input_size = input_dim_ * sizeof(float);
    hipMalloc(&d_input, input_size);
    hipMemcpy(d_input, input, input_size, hipMemcpyHostToDevice);
    hipDeviceSynchronize();


    float *d_temp;

    // forward
    d_temp = matrixMul(weights_[0], d_input, hidden_dim_[0], input_dim_, input_dim_, 1);
    d_temp = matrixAdd(d_temp, biases_[0], hidden_dim_[0], 1);
    d_temp = elu(d_temp, 1, hidden_dim_[0]);

    for (int i = 1; i < hidden_dim_.size(); i++)
    {
        d_temp = matrixMul(weights_[i], d_temp, hidden_dim_[i], hidden_dim_[i - 1], hidden_dim_[i - 1], 1);
        d_temp = matrixAdd(d_temp, biases_[i], hidden_dim_[i], 1);
        d_temp = elu(d_temp, 1, hidden_dim_[i]);
    }

    d_temp = matrixMul(weights_[hidden_dim_.size()], d_temp, output_dim_, hidden_dim_[hidden_dim_.size() - 1], hidden_dim_[hidden_dim_.size() - 1], 1);
    d_temp = matrixAdd(d_temp, biases_[hidden_dim_.size()], output_dim_, 1);
    
    // memcpy output to host
    int output_size = output_dim_ * sizeof(float);
    hipMemcpy(output, d_temp, output_size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    //free memory
    hipFree(d_input);
    // hipFree(d_output);
    hipFree(d_temp);
}